#include "hip/hip_runtime.h"
#include "CudaKernels.h"
#include <hip/hip_runtime.h>
#include <cuda_d3d11_interop.h>
#include "Logger.h"

/**
 * @brief CUDA kernel to preprocess a raw image for inference.
 *
 * This kernel is executed by a grid of thread blocks. Each thread is responsible
 * for computing the value of a single pixel in the destination (output) tensor.
 * It maps the destination pixel back to the source image, applies letterbox
 * padding if necessary, normalizes the pixel value, and writes it to the
 * correct position in the CHW-formatted output buffer.
 */
__global__ void PreprocessKernel(hipTextureObject_t inputTexture, float* output, int img_w, int img_h, int output_w, int output_h)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= output_w || y >= output_h) {
        return;
    }

    float scale = min(static_cast<float>(output_w) / img_w, static_cast<float>(output_h) / img_h);
    int new_w = static_cast<int>(img_w * scale);
    int new_h = static_cast<int>(img_h * scale);
    int pad_x = (output_w - new_w) / 2;
    int pad_y = (output_h - new_h) / 2;

    int input_x = (x - pad_x) / scale;
    int input_y = (y - pad_y) / scale;

    float r_norm = 0.5f, g_norm = 0.5f, b_norm = 0.5f; // Default to padding color (grey)

    if (x >= pad_x && x < pad_x + new_w && y >= pad_y && y < pad_y + new_h) {
        // tex2D with hipReadModeNormalizedFloat gives us BGRA values in the [0.0, 1.0] range.
        float4 pixel = tex2D<float4>(inputTexture, (input_x + 0.5f) / img_w, (input_y + 0.5f) / img_h);
        
        // No need to divide by 255.0f again. Just swizzle from BGRA to RGB for the model.
        r_norm = pixel.z; // R channel
        g_norm = pixel.y; // G channel
        b_norm = pixel.x; // B channel
    }

    // NCHW format
    output[0 * output_h * output_w + y * output_w + x] = r_norm; // R channel
    output[1 * output_h * output_w + y * output_w + x] = g_norm; // G channel
    output[2 * output_h * output_w + y * output_w + x] = b_norm; // B channel
}

bool PreprocessD3D11Texture(ID3D11Texture2D* pTexture, float* d_processed_output, int texture_w, int texture_h, int output_w, int output_h, hipGraphicsResource** ppCudaResource)
{
    hipError_t err;

    // Unregister previous resource if it exists
    if (*ppCudaResource != nullptr) {
        hipGraphicsUnregisterResource(*ppCudaResource);
        *ppCudaResource = nullptr;
    }
    
    // Register the D3D11 texture as a CUDA graphics resource
    err = cudaGraphicsD3D11RegisterResource(ppCudaResource, pTexture, hipGraphicsRegisterFlagsNone);
    if (err != hipSuccess) {
        Logger::GetInstance().Log("cudaGraphicsD3D11RegisterResource failed: %s", hipGetErrorString(err));
        return false;
    }

    // Map the resource to access it from CUDA
    err = hipGraphicsMapResources(1, ppCudaResource, 0);
    if (err != hipSuccess) {
        Logger::GetInstance().Log("hipGraphicsMapResources failed: %s", hipGetErrorString(err));
        return false;
    }

    hipArray* cuArray;
    err = hipGraphicsSubResourceGetMappedArray(&cuArray, *ppCudaResource, 0, 0);
    if (err != hipSuccess) {
        Logger::GetInstance().Log("hipGraphicsSubResourceGetMappedArray failed: %s", hipGetErrorString(err));
        return false;
    }

    // Create a texture object to read from the CUDA array
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = cuArray;

    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.readMode = hipReadModeNormalizedFloat;
    texDesc.normalizedCoords = 1;

    hipTextureObject_t texObj = 0;
    err = hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);
     if (err != hipSuccess) {
        Logger::GetInstance().Log("hipCreateTextureObject failed: %s", hipGetErrorString(err));
        hipGraphicsUnmapResources(1, ppCudaResource, 0);
        return false;
    }

    dim3 block(16, 16);
    dim3 grid((output_w + block.x - 1) / block.x, (output_h + block.y - 1) / block.y);

    PreprocessKernel<<<grid, block>>>(texObj, d_processed_output, texture_w, texture_h, output_w, output_h);

    hipDestroyTextureObject(texObj);
    hipGraphicsUnmapResources(1, ppCudaResource, 0);
    
    // The resource should be unregistered when it's no longer needed, typically on cleanup.
    // We leave it registered for the lifetime of the texture.

    return true;
}

float* AllocatePreprocessedBuffer(int width, int height) {
    float* d_buffer;
    hipMalloc(&d_buffer, 3 * width * height * sizeof(float));
    return d_buffer;
} 